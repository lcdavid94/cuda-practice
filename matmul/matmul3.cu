
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
const int DIM = 128;
const int N = DIM*DIM;
const int CUDA_DIM = 128;
const int MOD = 5;

void printMat(int *m) {
#ifndef NO_PRINT
    for (int i=0; i<N; ++i) {
        printf("%4d", m[i]);
        if (i%DIM==DIM-1) {
            printf("\n");
        }
    }
#endif
}

int *cache;

struct mat {
    int *p;
    mat(int n) {
        p = cache;
        cache += n*DIM;
    }
    __device__ int& operator[](int n) {
        return p[n];
    }
    int* operator+(int n) {
        return p+n;
    }
    ~mat() {
        cache = p; 
    }
};

struct addOrSubInfo {
    int *a, *b, *c;
    bool add;
    addOrSubInfo(int *a, int *b, int *c, bool add):
        a(a), b(b), c(c), add(add) {}
    addOrSubInfo(): a(NULL) {}
};

struct infoSet {
    addOrSubInfo infoSet[11];
};

struct MulInfo {
	int *a, *b, *c;
	MulInfo(int *a, int *b, int *c):
	    a(a), b(b), c(c) {}
	MulInfo() {}
};

struct MulInfoSet {
	MulInfo infoSet[7];
};

__global__ void matAddOrSub(infoSet addOrSubInfoSet) {
    int x = blockIdx.x, y = threadIdx.x;
    int offset = x*DIM + y;
    addOrSubInfo *info = addOrSubInfoSet.infoSet;
    while (info->a) {
        info->c[offset] = info->add ? 
            info->a[offset]+info->b[offset]:info->a[offset]-info->b[offset];
        ++info;
    }
}

__global__ void matMulCuda(MulInfoSet mulInfoSet) {
	int idx = blockIdx.x;
    int x = blockIdx.y, y = threadIdx.x;
    int offset = x*DIM + y;
    int dim = blockDim.x;
    int *a = mulInfoSet.infoSet[idx].a, *b = mulInfoSet.infoSet[idx].b,
        *c = mulInfoSet.infoSet[idx].c;
    c[offset] = 0;
    for (int i=0; i<dim; ++i) {
        c[offset] += a[x*DIM+i]*b[i*DIM+y];
    }
}

void matMul(int *a, int *b, int *c, int dim) {
    if (dim <= CUDA_DIM) {
		dim3 d(1, dim);
		MulInfoSet mulInfoSet;
		mulInfoSet.infoSet[0] = MulInfo(a, b, c);
        matMulCuda<<<d, dim>>>(mulInfoSet);
        return;
    }
    mat p1(dim), p2(dim), p3(dim);
    int half = dim/2;
    int *a11=a, *a12=a+half, *a21=a+half*DIM, *a22=a+half*DIM+half;
    int *b11=b, *b12=b+half, *b21=b+half*DIM, *b22=b+half*DIM+half;
    int *s1=p1+0, *s2=p1+half, *s3=p1+half*DIM, *s4=p1+half*DIM+half;
    int *s5=p2+0, *s6=p2+half, *s7=p2+half*DIM, *s8=p2+half*DIM+half;
    int *s9=p3+0, *s10=p3+half;

    infoSet addOrSubInfoSet;
    addOrSubInfoSet.infoSet[0] = addOrSubInfo(a11, a22, s1, true);
    addOrSubInfoSet.infoSet[1] = addOrSubInfo(b11, b22, s2, true);
    addOrSubInfoSet.infoSet[2] = addOrSubInfo(a21, a22, s3, true);
    addOrSubInfoSet.infoSet[3] = addOrSubInfo(b12, b22, s4, false);
    addOrSubInfoSet.infoSet[4] = addOrSubInfo(b21, b11, s5, false);
    addOrSubInfoSet.infoSet[5] = addOrSubInfo(a11, a12, s6, true);
    addOrSubInfoSet.infoSet[6] = addOrSubInfo(a21, a11, s7, false);
    addOrSubInfoSet.infoSet[7] = addOrSubInfo(b11, b12, s8, true);
    addOrSubInfoSet.infoSet[8] = addOrSubInfo(a12, a22, s9, false);
    addOrSubInfoSet.infoSet[9] = addOrSubInfo(b21, b22, s10, true);
    addOrSubInfoSet.infoSet[10] = addOrSubInfo();
    matAddOrSub<<<half, half>>>(addOrSubInfoSet);
    mat q1(dim), q2(dim);
    int *m1=q1+0, *m2=q1+half, *m3=q1+half*DIM, *m4=q1+half*DIM+half;
    int *m5=q2+0, *m6=q2+half, *m7=q2+half*DIM;
    if (dim > CUDA_DIM*2) {
        matMul(s1, s2, m1, half);
        matMul(s3, b11, m2, half);
        matMul(a11, s4, m3, half);
        matMul(a22, s5, m4, half);
        matMul(s6, b22, m5, half);
        matMul(s7, s8, m6, half);
        matMul(s9, s10, m7, half);
    } else {
       MulInfoSet mulInfoSet;
       mulInfoSet.infoSet[0] = MulInfo(s1, s2, m1);
       mulInfoSet.infoSet[1] = MulInfo(s3, b11, m2);
       mulInfoSet.infoSet[2] = MulInfo(a11, s4, m3);
       mulInfoSet.infoSet[3] = MulInfo(a22, s5, m4);
       mulInfoSet.infoSet[4] = MulInfo(s6, b22, m5);
       mulInfoSet.infoSet[5] = MulInfo(s7, s8, m6);
       mulInfoSet.infoSet[6] = MulInfo(s9, s10, m7);
       dim3 d(7, half);
       matMulCuda<<<d, half>>>(mulInfoSet);
    }
    int *c11=c, *c12=c+half, *c21=c+half*DIM, *c22=c+half*DIM+half;
    addOrSubInfoSet.infoSet[0] = addOrSubInfo(m1, m4, c11, true);
    addOrSubInfoSet.infoSet[1] = addOrSubInfo(c11, m5, c11, false);
    addOrSubInfoSet.infoSet[2] = addOrSubInfo(c11, m7, c11, true);
    addOrSubInfoSet.infoSet[3] = addOrSubInfo(m3, m5, c12, true);
    addOrSubInfoSet.infoSet[4] = addOrSubInfo(m2, m4, c21, true);
    addOrSubInfoSet.infoSet[5] = addOrSubInfo(m1, m2, c22, false);
    addOrSubInfoSet.infoSet[6] = addOrSubInfo(c22, m3, c22, true);
    addOrSubInfoSet.infoSet[7] = addOrSubInfo(c22, m6, c22, true);
    addOrSubInfoSet.infoSet[8] = addOrSubInfo();
    matAddOrSub<<<half, half>>>(addOrSubInfoSet);
}

int main() {
    int a[N], b[N], c[N];
    for (int i=0; i<N; ++i) {
        a[i] = rand()%MOD;
        b[i] = rand()%MOD;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc(&cache, sizeof(int[N*5]));
    int *devA, *devB, *devC;
    hipMalloc(&devA, sizeof(int[N]));
    hipMalloc(&devB, sizeof(int[N]));
    hipMalloc(&devC, sizeof(int[N]));
    hipMemcpy(devA, a, sizeof(int[N]), hipMemcpyHostToDevice);
    hipMemcpy(devB, b, sizeof(int[N]), hipMemcpyHostToDevice);
    matMul(devA, devB, devC, DIM);
    hipMemcpy(c, devC, sizeof(int[N]), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    printf("a:\n");
    printMat(a);
    printf("\nb:\n");
    printMat(b);
    printf("\nc:\n");
    printMat(c);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\nTime taken for matrix multiplication: %.3f ms", elapsedTime);

    int ans[N];
    for (int i=0; i<DIM; ++i) {
        for (int j=0; j<DIM; ++j) {
            int offset = i*DIM+j;
            ans[offset] = 0;
            for (int k=0; k<DIM; ++k) {
                ans[offset] += a[i*DIM+k]*b[k*DIM+j];
            }
        }
    }
    try {
        for (int i=0; i<N; ++i) {
            if (ans[i]!=c[i]) {
                throw 1;
            }
        }
        printf("\nSuccess!\n");
    } catch (int) {
        printf("\nFailed\n");
    }

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipFree(cache);
}
